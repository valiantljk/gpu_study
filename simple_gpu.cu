/* from: https://devblogs.nvidia.com/parallelforall/even-easier-introduction-cuda/
 * Jialin Liu
 * Simple starting cpp cuda program
 * Jun 24 2017, Saturday, 2:09pm
 * Compile and test on Maeve, a 3GPU single node at NERSC, LBNL, CA. 
*/



#include <hip/hip_runtime.h>
#include<iostream>
#include<math.h>
using namespace std;

//CUDA kernel functions to add the elements of two arrays
__global__
void add (int n, float *x, float * y){
  for (int i=0;i<n;i++){
      y[i] = x[i] + y[i];
  }
}

int main(void)
{
  int N= 1<<20; //1 million elements
  //float * x= new float[N];
  //float * y= new float[N];
  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
  clock_t t;
  //Initialize x and y arrays on the host
  for (int i=0; i<N; i++){
   x[i] =1.5f;
   y[i] =2.3f;
  }
  
  //run kernel on 1M elements on the CPU
  t = clock();
  //add(N, x, y);
  add<<<1, 1>>>(N, x, y);
  t = clock() -t;
  //cout<<format("%f seconds")%((float)t/CLOCKS_PER_SEC)<<endl;
  cout <<(float)t/CLOCKS_PER_SEC<<" seconds"<<endl;
  //Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  float maxError = 0.0f;
  for (int i =0;i <N;i ++)
   maxError =fmax(maxError, fabs(y[i]-3.8f));
  cout <<"Max error: "<<maxError <<endl;

  //delete [] x;
  //delete [] y;
  hipFree(x);
  hipFree(y);
  return 0;
}
